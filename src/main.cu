#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>

#include "dgemm.cuh"
#include "cuda_common.cuh"

//  n=64까지만 동작함.
void BM_DGEMM_CUDA01(benchmark::State& state) {
    int n = state.range(0);
    size_t size = sizeof(double) * n * n;

    double* h_A = (double*)malloc(size);
    double* h_B = (double*)malloc(size);
    double* h_C = (double*)malloc(size);

    double *d_A; hipMallocManaged(&d_A, size);
    double *d_B; hipMallocManaged(&d_B, size);
    double *d_C; hipMallocManaged(&d_C, size);

    srand((unsigned int)time(NULL));
    initializeDM(h_A, n);
    initializeDM(h_B, n);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    for (auto _ : state){
        dgemm_cuda_01<<<1,1>>>(d_A, d_B, d_C, n);
    }

    hipFree(d_A); free(h_A);
    hipFree(d_B); free(h_B);
    hipFree(d_C); free(h_C);
}

void BM_DGEMM_CUDA02(benchmark::State& state) {
    int n = state.range(0);
    size_t size = sizeof(double) * n * n;

    double* h_A = (double*)malloc(size);
    double* h_B = (double*)malloc(size);
    double* h_C = (double*)malloc(size);

    double *d_A; hipMalloc(&d_A, size);
    double *d_B; hipMalloc(&d_B, size);
    double *d_C; hipMalloc(&d_C, size);

    // int threadsPerBlock = 256;
    // int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    dim3 threads(1024, 1024);
    dim3 grid(ceil(n / (float)threads.x), ceil(n / (float)threads.y));

    srand((unsigned int)time(NULL));
    initializeDM(h_A, n);
    initializeDM(h_B, n);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    for (auto _ : state){
        cuda_event_timer raii{state};
        dgemm_cuda_02<<<grid,threads>>>(d_A, d_B, d_C, n);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
}


//  use unified memory
//  n=2^14에서 왜 Segmentatioin Fault가 나는가?
void BM_DGEMM_CUDA03(benchmark::State& state) {
    int n = state.range(0);
    size_t size = sizeof(double) * n * n;

    double* a; hipMallocManaged(&a, size);
    double* b; hipMallocManaged(&b, size);
    double* c; hipMallocManaged(&c, size);

    dim3 threads(1024, 1024);
    dim3 grid(ceil(n / (float)threads.x), ceil(n / (float)threads.y));

    srand((unsigned int)time(NULL));
    initializeDM(a, n);
    initializeDM(b, n);

    for (auto _ : state){
        cuda_event_timer raii{state};
        dgemm_cuda_02<<<grid,threads>>>(a, b, c, n);
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);
}


//  for Geforce 3080 ti
//  core number: 10240
/*
    Maximum Texture Dimension Size (x,y,z)
    1D=(131072), 2D=(131072, 65536), 3D=(16384, 16384, 16384)
*/
void BM_DGEMM_CUDA04(benchmark::State& state) {
    int n = state.range(0);
    size_t size = sizeof(double) * n * n;

    double* a; hipMallocManaged(&a, size);
    double* b; hipMallocManaged(&b, size);
    double* c; hipMallocManaged(&c, size);

    dim3 threads(1024, 1024);
    dim3 grid(ceil(n / (float)threads.x), ceil(n / (float)threads.y));

    srand((unsigned int)time(NULL));
    initializeDM(a, n);
    initializeDM(b, n);

    for (auto _ : state){
        cuda_event_timer raii{state};
        dgemm_cuda_02<<<grid,threads>>>(a, b, c, n);
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);
}


void BM_SGEMM_CUDA01(benchmark::State& state) {
    int n = state.range(0);
    size_t size = sizeof(double) * n * n;

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    float *d_A; hipMallocManaged(&d_A, size);
    float *d_B; hipMallocManaged(&d_B, size);
    float *d_C; hipMallocManaged(&d_C, size);

    dim3 threads(1024, 1024);
    dim3 grid(ceil(n / (float)threads.x), ceil(n / (float)threads.y));

    srand((unsigned int)time(NULL));
    initializeFM(h_A, n);
    initializeFM(h_B, n);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    for (auto _ : state){
        cuda_event_timer raii{state};
        sgemm_cuda_01<<<grid,threads>>>(d_A, d_B, d_C, n);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
}