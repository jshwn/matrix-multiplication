
#include <hip/hip_runtime.h>
//  with 1 thread 1 block
__global__ void dgemm_cuda_01(double *a, double *b, double *c, int N) {
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            for(int k=0; k<N; k++){
                *(c+i*N+j) += (*(a+i*N+k)) * (*(b+k*N+j));
            }
        }
    }
}

//  standard
__global__ void dgemm_cuda_02(double *a, double *b, double *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row<N && col<N){
        double v = 0.0;
        for(int i=0;i<N;i++){
            v += *(a+row*N+i) + *(b+i*N+col);
        }
        *(c+row*N+col) = v;
    }
}

//  cache friendly
__global__ void dgemm_cuda_03(double *a, double *b, double *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row<N && col<N){
        double v = 0.0;
        for(int i=0;i<N;i++){
            v += *(a+row*N+i) + *(b+i*N+col);
        }
        *(c+row*N+col) = v;
    }
}


__global__ void sgemm_cuda_01(float* a, float* b, float* c, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row<N && col<N){
        double v = 0.0;
        for(int i=0;i<N;i++){
            v += *(a+row*N+i) + *(b+i*N+col);
        }
        *(c+row*N+col) = v;
    }
};